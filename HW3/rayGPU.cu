#include "hip/hip_runtime.h"
// Kelby Hubbard
// CS F441
// Homework 3
// Problem 5


// This is a simple ray tracer that shoots rays top down toward randomly
// generates spheres and draws the sphere in a random color based on where
// the ray hits it.

#include "FreeImage.h"
#include "stdio.h"

#define DIM 2048
#define rnd(x) (x * rand() / RAND_MAX)
#define INF 2e10f

struct Sphere {
    float   r,b,g;
    float   radius;
    float   x,y,z;
    // Tells us if a ray hits the sphere; return the
    // depth of the hit, or -infinity if the ray misses the sphere
    __device__ float hit( float ox, float oy, float *n ) 
    {
        float dx = ox - x;
        float dy = oy - y;
        if (dx*dx + dy*dy < radius*radius)
        {
            float dz = sqrtf( radius*radius - dx*dx - dy*dy );
            *n = dz / sqrtf( radius * radius );
            return dz + z;
        }
        return -INF;
    }
};

#define SPHERES 80

// Loops through each pixel in the image (represented by arrays of
// red, green, and blue) and then for each pixel checks if a ray from
// top down hits one of the randomly generated spheres.
// If so, calculate a shade of color based on where the ray hits it.
__global__ void drawSpheres(Sphere spheres[], char *red, char *green, char *blue)
{
  int x = blockIdx.x;
  int y = blockIdx.y;
	float   ox = (x - DIM/2);
	float   oy = (y - DIM/2);

	float   r=0, g=0, b=0;
	float   maxz = -INF;
	for(int i=0; i<SPHERES; i++)
 	{
        	float   n;
        	float   t = spheres[i].hit( ox, oy, &n );
        	if (t > maxz)
		      {
			      // Scale RGB color based on z depth of sphere
            float fscale = n;
          	r = spheres[i].r * fscale;
          	g = spheres[i].g * fscale;
          	b = spheres[i].b * fscale;
          	maxz = t;
        	}
        }
  int offset = x + y * DIM;
  red[offset] = (char) (r * 255);
  green[offset] = (char) (g * 255);
  blue[offset] = (char) (b * 255);
}

int main()
{
  FreeImage_Initialise();
  atexit(FreeImage_DeInitialise);
  FIBITMAP * bitmap = FreeImage_Allocate(DIM, DIM, 24);
  srand(time(NULL));

  char *red, *green, *blue;
  char *dev_red, *dev_green, *dev_blue;

  // Dynamically create enough memory for DIM * DIM array of char.
  // By making these dynamic rather than auto (e.g. char red[DIM][DIM])
  // we can make them much bigger since they are allocated off the heap
  hipMalloc((void**)&dev_red, sizeof(char)*DIM*DIM);
  hipMalloc((void**)&dev_green, sizeof(char)*DIM*DIM);
  hipMalloc((void**)&dev_blue, sizeof(char)*DIM*DIM);
  red = new char[DIM*DIM];
  green = new char[DIM*DIM];
  blue = new char[DIM*DIM];


  // Create random spheres at different coordinates, colors, radius
  Sphere spheres[SPHERES];
  for (int i = 0; i<SPHERES; i++)
  {
        spheres[i].r = rnd( 1.0f );
        spheres[i].g = rnd( 1.0f );
        spheres[i].b = rnd( 1.0f );
        spheres[i].x = rnd( (float) DIM ) - (DIM/2.0);
        spheres[i].y = rnd( (float) DIM ) - (DIM/2.0);
        spheres[i].z = rnd( (float) DIM ) - (DIM/2.0);
        spheres[i].radius = rnd( 200.0f ) + 40;
  }
  Sphere dev_spheres[SPHERES];
  hipMalloc((void**)&dev_spheres, sizeof(Sphere)*SPHERES);
  hipMemcpy(dev_spheres, spheres, sizeof(Sphere)*SPHERES, hipMemcpyHostToDevice);

  // Copy the data to the GPU
  hipMemcpy(dev_red, red, DIM*DIM*sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(dev_green, green, DIM*DIM*sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(dev_blue, blue, DIM*DIM*sizeof(char), hipMemcpyHostToDevice);
  // Create a kernel to draw the spheres
  dim3 grid(DIM, DIM); // Grid size
  drawSpheres <<< grid, 1 >>> (dev_spheres, dev_red, dev_green, dev_blue);
  // Copy the data back to the host
  hipMemcpy(red, dev_red, sizeof(char)*DIM*DIM, hipMemcpyDeviceToHost);
  hipMemcpy(green, dev_green, sizeof(char)*DIM*DIM, hipMemcpyDeviceToHost);
  hipMemcpy(blue, dev_blue, sizeof(char)*DIM*DIM, hipMemcpyDeviceToHost);
  
  RGBQUAD color;
  for (int i = 0; i < DIM; i++)
  {
    for (int j = 0; j < DIM; j++)
    {
      int index = j*DIM + i;
      color.rgbRed = red[index];
      color.rgbGreen = green[index];
      color.rgbBlue = blue[index];
      FreeImage_SetPixelColor(bitmap, i, j, &color);
    }
  }
	
  FreeImage_Save(FIF_PNG, bitmap, "ray.png", 0);
  FreeImage_Unload(bitmap);
  hipFree(dev_red);
  hipFree(dev_green);
  hipFree(dev_blue);
  delete(red);
  delete(green);
  delete(blue);

  return 0;
}